#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <fstream>
#include <iomanip>


// The cuda kernel
__global__ void quamsim_kernel(const float *qstates,const float *qbit_gate, float *q_out, int n_size,int q_index ) {


  int i = blockIdx.x* blockDim.x + threadIdx.x;

  int i_opp = i ^ (1<<q_index);

  if(i<n_size){
    if((i_opp & (1<<q_index))){
      q_out[i] = qbit_gate[0] * qstates[i] + qbit_gate[1] * qstates[i_opp];
      q_out[i_opp] = qbit_gate[2] * qstates[i] + qbit_gate[3] * qstates[i_opp];
    }
  }
}


int main(int argc, char *argv[]) {

  // Read the inputs from command line

  char *trace_file;
  trace_file = argv[1];

  std::ifstream file(trace_file);

  float gate[4];

  for(int i = 0; i<4; i++){
      
          file >> gate[i];
      
  }

  std::vector<float> states;
  
  float instate;
  while(file>>instate){
      states.push_back(instate);
  }

  int t;
  t = states.back();
  states.pop_back();

  int n = states.size();

  size_t size = n* sizeof(float);
  size_t size_gate = 4 * sizeof(float);
  // Allocate/move data using cudaMalloc and cudaMemCpy

  float *host_states = (float*) malloc(size);
  float *host_gate = (float*)malloc(size_gate);

  float *host_out = (float*)malloc(size);

  for (int i = 0; i < n; ++i)
  {
      host_states[i] = states[i];
  }

  for (int i = 0; i < 4; i++) 
  {
      host_gate[i]= gate[i];
  }

  //allocating the memory in GPU
  float *d_states, *d_gate, *d_out;

  hipMalloc((void**)&d_states, size);
  hipMalloc((void**)&d_gate, size_gate);
  hipMalloc((void**)&d_out, size);

  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  
  hipMemcpy(d_states, host_states, size, hipMemcpyHostToDevice);
  hipMemcpy(d_gate, host_gate, size_gate, hipMemcpyHostToDevice);



  int threadsPerBlock = 256;
  int blocksPerGrid = (n+threadsPerBlock-1/threadsPerBlock);

  // Launch the kernel
  hipEventRecord(start);
  quamsim_kernel<<<blocksPerGrid, threadsPerBlock >>>(d_states,d_gate,d_out,n, t);
  hipEventRecord(stop);
  // Print the output

  // Clean up the memory
  hipMemcpy(host_states, d_states, size, hipMemcpyDeviceToHost);
  hipMemcpy(host_gate, d_gate, size_gate, hipMemcpyDeviceToHost);
  hipMemcpy(host_out, d_out, size, hipMemcpyDeviceToHost);

//printing to be written

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout<< milliseconds << std::endl;
  for(int i = 0; i<n; i++){
    std::cout<< std::fixed << std::setprecision(3) << host_out[i] << std::endl;
  }
  hipFree(d_states);
  hipFree(d_gate);
  hipFree(d_out);

  free(host_states);
  free(host_gate);
  free(host_out);


  
  
  printf("Done\n");
  return 0;
}
